#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>

// D:\Visual Studio\VC\Tools\MSVC\14.29.30133\bin\Hostx86\x64
#define TYPE double
#define N (1000*1000*100)
#define THREDA_N 5000 //分为SIZE个部分
#define PROCESSOR 8
#define GPU
#define MUTITHREAD
#define INFO
/*
    para:
        len: 每个线程需要循环的次数，第n个线程计算 [n*len , n*len + len ];
*/
#ifdef GPU
__global__ void d_integration(int len, TYPE *d_result) {
  int ix = threadIdx.x + (blockIdx.x * blockDim.x);
  int iy = threadIdx.y + (blockIdx.y * blockDim.y);
  int tId = iy * (blockDim.x * gridDim.x) + ix;
  //printf("GPU: thread ID = %d  calculate [%d,%d]\n", tId, tId * len,tId * len + len );
  for (int i = tId * len; i < tId * len + len; i++) {
    d_result[tId] += (1.0 / N) * (4.0 / (1.0 + (pow((i + 0.5) / N, 2.0))));
  }
}
__global__ void d_arctan(int len, TYPE *d_result){
  int ix = threadIdx.x + (blockIdx.x * blockDim.x);
  int iy = threadIdx.y + (blockIdx.y * blockDim.y);
  int tId = iy * (blockDim.x * gridDim.x) + ix;
  int sign = 0;
  tId%2==0?sign = -1:sign = 1;
  for (int i = tId*len ; i < tId*len + len ; i++)
  {
      d_result[tId] += sign*(1/(2(i+1) -1));
  }
}
#endif

TYPE h_integration() {
  TYPE result = 0;
  for (int i = 0; i < N; i++) {
    result += (1.0 / N) * (4.0 / (1.0 + (pow((i + 0.5) / N, 2.0))));
  }
  return result;
}

int main() {
#ifdef GPU
  #ifdef INFO
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for(int i=0;i<deviceCount;i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printf("USING GPU device %d : %s \n",i,devProp.name);
        printf("Total memory : %zd MB \n",devProp.totalGlobalMem/(1024*1024));
        printf("SM : %d \n",devProp.multiProcessorCount);
        printf("Shared Memory for each block %zd \n",devProp.sharedMemPerBlock);
        printf("MaxBlock(%d,%d,%d) MaxGrid(%d,%d,%d) \n",devProp.maxThreadsDim[0],devProp.maxThreadsDim[1],devProp.maxThreadsDim[2],devProp.maxGridSize[0],devProp.maxGridSize[1],devProp.maxGridSize[2]);
        printf("Major compute capability %d \n",devProp.major);

    }
  #endif
  int sizePerThread = N / THREDA_N;
  int sizeOfResult = sizeof(TYPE) * THREDA_N;
  TYPE *h_result = (TYPE *)malloc(sizeOfResult);
  memset(h_result, 0, sizeOfResult);

  TYPE *d_result;
  hipError_t memallocError = hipMalloc((TYPE **)&d_result, sizeOfResult);
  printf("cuda : malloc %s \n", hipGetErrorString(memallocError));

  hipError_t memcpyError = hipMemcpy(d_result, h_result, sizeOfResult, hipMemcpyHostToDevice);
  printf("cuda : malloc %s \n", hipGetErrorString(memcpyError));

  dim3 block(500,2,1);
  dim3 grid(5,1,1);
  double dur;
  clock_t start,end;
  start = clock();
  d_integration<<<grid, block>>>(sizePerThread, d_result);

  hipMemcpy(h_result, d_result, sizeOfResult, hipMemcpyDeviceToHost);
  TYPE finResult = 0;
  for (int i = 0; i < THREDA_N; i++) {
   //printf("%f ", h_result[i]);
    finResult += h_result[i];
  }
  end = clock();
  dur = (double)(end - start);
  printf("CPU : Use Time:%f\n",(dur/CLOCKS_PER_SEC));
#else
  double dur;
  clock_t start,end;
  start = clock();
  TYPE finResult = h_integration();
  end = clock();
  dur = (double)(end - start);
  printf("CPU : Use Time:%f\n",(dur/CLOCKS_PER_SEC));
#endif

  printf("\n PI = %f \n ", finResult);
}