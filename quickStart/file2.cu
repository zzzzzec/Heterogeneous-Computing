
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex(void){
    //printf("(%d,%d,%d) blockIndex(%d,%d,%d) blockDim(%d,%d,%d) GridDim(%d,%d,%d) \n",threadIdx.x , threadIdx.y , threadIdx.z , blockIdx.x , blockIdx.y , blockIdx.z,blockDim.x,blockDim.y,blockDim.z ,gridDim.x,gridDim.y,gridDim.z);
    int ix = threadIdx.x + (blockIdx.x * blockDim.x);
    int iy = threadIdx.y + (blockIdx.y * blockDim.y);
    int id = iy * (blockDim.x * gridDim.x) + ix;
    printf("thread id = %d \n",id);
}

int main(){
    int nElem = 6 ;
    dim3 block(4,2,1); 
    //dim3 grid((nElem + block.x - 1)/block.x);
    dim3 grid(2,3,1);
    printf("grid.x = %d grid.y = %d grid.z = %d \n",grid.x,grid.y,grid.z);
    printf("block.x = %d block.y = %d block.z = %d \n",block.x,block.y,block.z);
    //注意！！！ 是<<<grid,block>>>不要弄反了！！！
    checkIndex <<<grid,block>>> ();
    exit(-1);
}