#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>
#include <time.h>

#define ARRAY_LEN (1000)
#define ARRAY_SIZE(type) (sizeof(type)*ARRAY_LEN)

__global__ void Summary(float* d_a , float* d_b , float* d_result){
    int i = threadIdx.x;
    {
       d_result[i] = d_a[i] + d_b[i];
    }
   
}

void display(float data[] , int len){
    for (int i = 0; i < len; i++)
    {
        printf("%f ",data[i]);
    }
    printf("\n");
}

void set_memory(float data[] , int len , float number){
    for (int i = 0; i < len; i++)
    {
        data[i] = number;
    }
    
}

int main(){
    float* h_a;
    float* h_b;
    float* h_recv_result;
    h_a = (float*)malloc(sizeof(float) * ARRAY_LEN);
    h_b = (float*)malloc(sizeof(float) * ARRAY_LEN);
    h_recv_result = (float*)malloc(sizeof(float) * ARRAY_LEN);
    //memset(h_a , 0 , ARRAY_SIZE);
    //memset(h_b , 0 , ARRAY_SIZE);
    set_memory(h_a , ARRAY_LEN , 10.0);
    set_memory(h_b , ARRAY_LEN , 20.0);

    float* d_a;
    float* d_b;
    float* d_c;
    hipError_t d_memError;
    const char* d_memError_char;
    d_memError = hipMalloc((float**)&d_a , ARRAY_SIZE(float) );
    d_memError_char = hipGetErrorString(d_memError);
    printf("%s\n",d_memError_char);
    d_memError = hipMalloc((float**)&d_b , ARRAY_SIZE(float) );
    d_memError_char = hipGetErrorString(d_memError);
    printf("%s\n",d_memError_char);
    d_memError = hipMalloc((float**)&d_c , ARRAY_SIZE(float) );
    d_memError_char = hipGetErrorString(d_memError);
    printf("%s\n",d_memError_char);

    hipMemcpy(d_a , h_a , ARRAY_SIZE(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_b , h_b , ARRAY_SIZE(float) , hipMemcpyHostToDevice);

    dim3 block(ARRAY_LEN);
    dim3 grid(ARRAY_LEN/block.x);
    double iStart = clock();
    Summary<<<grid,block>>>(d_a , d_b , d_c);   
    printf("execution configuration <<<%d,%d>>> \n",grid.x,block.x);
    hipMemcpy(h_recv_result , d_c , ARRAY_SIZE(float) , hipMemcpyDeviceToHost);
    printf("total time %f \n",iStart - clock());
    for (int i = 0; i < ARRAY_LEN ; i++)
    {
        printf("%f ",h_recv_result[i]);
    }
    

}