#include<stdio.h>

int main(){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(deviceProp,0);
    printf("name %s \n",deviceProp.name);
    printf("SM = %d\n",deviceProp.multiProcessorCount);
    printf("shared memory = %d\n",deviceProp.sharedMemPerBlock);
    
}