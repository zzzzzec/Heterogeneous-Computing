#include "hip/hip_runtime.h"
/*关于此文件的注释*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include ""

//最小调度单位 warp 一个 warp有32个线程
// warp之间都是同步的，__syncthreads（）可以保证一个block内的warp是同步的
// 有点像 barrier ，所有的线程都执行到 __syncthreads 之后才会继续向前运行
//"D:\Visual Studio\VC\Tools\MSVC\14.29.30133\bin\Hostx86\x64"
#define N 512
#define TIMES 10
#define MATRIX_SIZE N*N*(sizeof(TYPE))
#define WARPNUMBER N/32
#define FAST

#define TYPE double

TYPE* h_A;//矩阵指针
TYPE* h_result;
//TYPE* temp;  

typedef struct
{
    int available;
    double *result;
} list;

void set_list(list *result_list, int times)
{
    for (int i = 0; i < 32; i++)
    {
        if ((times >> i) & 0x00000001)
        {
            result_list[i].available = 1;
        }
    }
}

void mem_check(TYPE* base, float min, float max) {
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            assert(*(base + i * N + j) < max && *(base + i * N + j) > min);
        }
    }
}
double* getAddr(double* base, int x, int y)
{
    return (base + (x * N) + y);
}

double getValue(double* base, int x, int y)
{
    return *(base + (x * N) + y);
}

/*
    矩阵乘法：左边的x行乘以右边的y
    retrun 单个元素结果
*/
double muti_matrix_template(double* left, double* right, int x, int y)
{
    double result = 0.0;
    for (int i = 0; i < N; i++)
    {
        //assert(getValue(left, x, i) < 600);
        result += (getValue(left, x, i) * getValue(right, i, y));
    }
    return result;
}

void muti_matrix(double* left, double* right, double* result)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            *(getAddr(result, i, j)) = muti_matrix_template(left, right, i, j);
        }
    }
}

__device__ TYPE* d_getAddr(TYPE* base , int x , int y){
    return (base + (x * N) + y); 
}

__device__ TYPE d_getValue(TYPE* base , int x , int y){
    return *(base + (x * N) + y);
}

void init(TYPE* base , TYPE value){
    TYPE* index = base;
    for (int i = 0; i < N*N ; i++)
    {
        *(index) = value;
        index += 1;
    }
}

__device__ TYPE d_syncCaculate_template(TYPE* left , TYPE* right , int raw , int column){
    TYPE result = 0.0;
    for (int i = 0; i < N ; i++)
    {
        result += (d_getValue(left,raw,i) * d_getValue(right,i,column));   
    }
    return result;
}

__device__ TYPE d_syncCaculate_template_shared(TYPE* left , TYPE* shared_elm , int raw , int column){
    TYPE result = 0.0;
    for (int i = 0; i < N ; i++)
    {
        result += (d_getValue(left,raw,i) * *shared_elm );   
    }
    return result;
}

__global__ void syncCaculate(TYPE* d_A , TYPE* d_result , TYPE* d_temp , int rawPerThread , int times){
    //再考虑一下memcpy的位置，不能一个线程就把所有的memroy都cpoy完了
    //每个线程只CPOY自己的那一步分就好
    int tid = threadIdx.x; //+ (blockIdx.x*blockDim.x);
    //printf("%d %d %d %d \n",tid,threadIdx.x,blockDim.x,blockIdx.x);
    int start = tid * rawPerThread;
    int end = tid * rawPerThread + rawPerThread;
    int startMemOffset = (start * N);
    int copySize = (rawPerThread * N)*(sizeof(TYPE));

    //printf("thread %d : [%d,%d] startMemOffset = %d copySize = %d\n",tid,start,end-1,startMemOffset,copySize);
    
    memcpy(d_result + startMemOffset , d_A + startMemOffset, copySize);
    if(times == 1){
        return ;
    }
    memcpy(d_temp + startMemOffset , d_A + startMemOffset, copySize);
    for (int i = 0; i < times - 1 ; i++)
    {
        for (int raw = start ; raw < end ; raw ++)
        {
            for (int column = 0; column < N; column++)
            {
                *(d_getAddr(d_result, raw, column)) = d_syncCaculate_template(d_temp, d_A, raw, column);
            }
        }
        //覆盖问题，每个线程只复制自己那一部分内存
        memcpy(d_temp + startMemOffset , d_result + startMemOffset, copySize);
        //if(threadIdx.x == 0){
           //printf("ROUND=%d = %f\n",i,d_result[0]);
        //}
        __syncthreads();    
    }    

    //printf("thread BLOCK=(%d,%d,%d) GRID=(%d,%d,%d) : down \n",threadIdx.x,threadIdx.y,threadIdx.z,blockIdx.x,blockIdx.y,blockIdx.z);
    //printf("NOT BE THERE \n");
}

__global__ void synccalculate_fast(TYPE* base, int rawPerThread){
    int tid = threadIdx.x; //+ (blockIdx.x*blockDim.x);
    //printf("%d %d %d %d \n",tid,threadIdx.x,blockDim.x,blockIdx.x);
    int start = tid * rawPerThread;
    int end = tid * rawPerThread + rawPerThread;
    int startMemOffset = (start * N);
    int copySize = (rawPerThread * N)*(sizeof(TYPE));
    TYPE *temp = (TYPE *)malloc(copySize);

    for (int raw = start ; raw < end ; raw ++)
    {
        for (int column = 0; column < N; column++)
        {
                *(d_getAddr(temp, raw - start, column)) = d_syncCaculate_template(base, base, raw, column);
        }
    }
    __syncthreads();
    memcpy(base + startMemOffset, temp, copySize);
    __syncthreads();
}

int main(){
    h_A = (TYPE*)malloc(MATRIX_SIZE);
    h_result = (TYPE*)malloc(MATRIX_SIZE);
    init(h_A,1.01);
    init(h_result,0.0);

    int times = TIMES;

    int threadNumber = 512;
    int blockNumber = 1;
    dim3 block(threadNumber,1,1);
    dim3 grid(blockNumber,1,1);
    //int rawPerThread = N/(threadNumber*blockNumber);
    int rawPerThread = 1;
    printf("BLOCK =  (%d,%d,%d) GRID = (%d,%d,%d) \n",threadNumber,1,1,blockNumber,1,1);
    printf("rawPerThread = %d \n",rawPerThread);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);
    printf("name:%s\n",deviceProp.name);
    printf("SM = %d\n",deviceProp.multiProcessorCount);
    printf("share memory = %zd\n",deviceProp.sharedMemPerBlock);
  
#ifndef FAST
    TYPE* d_A;
    TYPE* d_result;
    TYPE* d_temp;
    hipMalloc( ((void**)(&d_A)) , MATRIX_SIZE);
    hipMalloc( ((void**)(&d_result)) , MATRIX_SIZE);
    hipMalloc( ((void**)(&d_temp)) , MATRIX_SIZE);

    hipMemcpy(d_A , h_A , MATRIX_SIZE ,hipMemcpyHostToDevice);

    syncCaculate<<<grid,block>>>(d_A , d_result , d_temp , rawPerThread , TIMES);

    hipMemcpy(h_result , d_result , MATRIX_SIZE , hipMemcpyDeviceToHost);
    printf("first_elm = %f\n",*h_result);

#else
    TYPE* d_base;
    hipMalloc(((void**)(&d_base)), MATRIX_SIZE);
    TYPE* temp = (TYPE*)malloc(MATRIX_SIZE);
    memcpy(temp, h_A, MATRIX_SIZE);
    
    list *result_list = (list *)malloc(sizeof(list) * (32));
    for (int i = 0; i < 32; i++)
    {
        result_list[i].available = 0;
        result_list[i].result = NULL;
    }

    set_list(result_list, times);

    int max_exp = floor(log2(times));

    for (int i = 1; i < max_exp + 1; i++)
    {
        hipMemcpy(d_base, temp, MATRIX_SIZE, hipMemcpyHostToDevice);
        synccalculate_fast << <grid, block >> > (d_base, rawPerThread);
        hipMemcpy(temp, d_base, MATRIX_SIZE, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if (result_list[i].available)
        {
            printf("add %d \n", i);
            double *add_to_result_list = (double *)malloc(MATRIX_SIZE);
            memcpy(add_to_result_list, temp, MATRIX_SIZE);
            result_list[i].result = add_to_result_list;
        }
    }

    //print_list(result_list);
    int flag = 1;
    double *fin_temp = (double *)malloc(MATRIX_SIZE);
    for (int i = 0; i < 32; i++)
    {
        if (result_list[i].available != 0)
        {
            if (flag)
            {
                memcpy(h_result, result_list[i].result, MATRIX_SIZE);
                memcpy(fin_temp, result_list[i].result, MATRIX_SIZE);
                flag = 0;
            }
            else
            {
                muti_matrix(fin_temp, result_list[i].result, h_result);
            }
            memcpy(fin_temp, h_result, MATRIX_SIZE);
        }
    }


#endif
    printf("first_elm = %f \n", h_result[0]);
    return 0;
}