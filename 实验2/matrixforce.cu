/*关于此文件的注释*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//最小调度单位 warp 一个 warp有32个线程
// warp之间都是同步的，__syncthreads（）可以保证一个block内的warp是同步的
// 有点像 barrier ，所有的线程都执行到 __syncthreads 之后才会继续向前运行
//"D:\Visual Studio\VC\Tools\MSVC\14.29.30133\bin\Hostx86\x64"
#define N 512
#define TIMES 32
#define MATRIX_SIZE N*N*(sizeof(TYPE))
#define WARPNUMBER N/32

#define TYPE double

TYPE* h_A;//矩阵指针
TYPE* h_result;
TYPE* temp;  

__device__ TYPE* getAddr(TYPE* base , int x , int y){
    return (base + (x * N) + y); 
}

__device__ TYPE getValue(TYPE* base , int x , int y){
    return *(base + (x * N) + y);
}

void init(TYPE* base , TYPE value){
    TYPE* index = base;
    for (int i = 0; i < N*N ; i++)
    {
        *(index) = value;
        index += 1;
    }
}

__device__ TYPE syncCaculate_template(TYPE* left , TYPE* right , int raw , int column){
    TYPE result = 0.0;
    for (int i = 0; i < N ; i++)
    {
        result += (getValue(left,raw,i) * getValue(right,i,column));   
    }
    return result;
}

__device__ TYPE syncCaculate_template_shared(TYPE* left , TYPE* shared_elm , int raw , int column){
    TYPE result = 0.0;
    for (int i = 0; i < N ; i++)
    {
        result += (getValue(left,raw,i) * *shared_elm );   
    }
    return result;
}

__global__ void syncCaculate(TYPE* d_A , TYPE* d_result , TYPE* d_temp , int rawPerThread , int times){
    //再考虑一下memcpy的位置，不能一个线程就把所有的memroy都cpoy完了
    //每个线程只CPOY自己的那一步分就好
    int tid = threadIdx.x; //+ (blockIdx.x*blockDim.x);
    //__shared__  TYPE d_A[N*N];
    __shared__ TYPE shared_elm ;
    if(tid == 1){
        shared_elm = 1.01;
    }
    //printf("%d %d %d %d \n",tid,threadIdx.x,blockDim.x,blockIdx.x);
    if(tid >= 512){
        return;
    }
    //printf("%d %d %d %d \n",tid,threadIdx.x,blockDim.x,blockIdx.x);
    int start = tid * rawPerThread;
    int end = tid * rawPerThread + rawPerThread;
    int startMemOffset = (start * N);
    int copySize = (rawPerThread * N)*(sizeof(TYPE));

    //printf("thread %d : [%d,%d] startMemOffset = %d copySize = %d\n",tid,start,end-1,startMemOffset,copySize);
    
    memcpy(d_result + startMemOffset , d_A + startMemOffset, copySize);
    if(times == 1){
        return ;
    }
    memcpy(d_temp + startMemOffset , d_A + startMemOffset, copySize);
    for (int i = 0; i < times - 1 ; i++)
    {
        for (int raw = start ; raw < end ; raw ++)
        {
            for (int column = 0; column < N; column++)
            {
                *(getAddr(d_result,raw,column)) = syncCaculate_template_shared(d_temp , &shared_elm , raw , column);
            }
        }
        //覆盖问题，每个线程只复制自己那一部分内存
        memcpy(d_temp + startMemOffset , d_result + startMemOffset, copySize);
        //if(threadIdx.x == 0){
           //printf("ROUND=%d = %f\n",i,d_result[0]);
        //}
        __syncthreads();    
    }    

    //printf("thread BLOCK=(%d,%d,%d) GRID=(%d,%d,%d) : down \n",threadIdx.x,threadIdx.y,threadIdx.z,blockIdx.x,blockIdx.y,blockIdx.z);
    //printf("NOT BE THERE \n");
}

int main(){
    h_A = (TYPE*)malloc(MATRIX_SIZE);
    h_result = (TYPE*)malloc(MATRIX_SIZE);
    init(h_A,1.01);
    init(h_result,0.0);

    TYPE* d_A;
    TYPE* d_result;
    TYPE* d_temp;
    hipMalloc( ((void**)(&d_A)) , MATRIX_SIZE);
    hipMalloc( ((void**)(&d_result)) , MATRIX_SIZE);
    hipMalloc( ((void**)(&d_temp)) , MATRIX_SIZE);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);
    printf("name:%s\n",deviceProp.name);
    printf("SM = %d\n",deviceProp.multiProcessorCount);
    printf("share memory = %zd\n",deviceProp.sharedMemPerBlock);
    hipMemcpy(d_A , h_A , MATRIX_SIZE ,hipMemcpyHostToDevice);

    int threadNumber = 512;
    int blockNumber = 1;
    dim3 block(threadNumber,1,1);
    dim3 grid(blockNumber,1,1);
    //int rawPerThread = N/(threadNumber*blockNumber);
    int rawPerThread = 1;
    printf("BLOCK =  (%d,%d,%d) GRID = (%d,%d,%d) \n",threadNumber,1,1,blockNumber,1,1);
    printf("rawPerThread = %d \n",rawPerThread);
    syncCaculate<<<grid,block>>>(d_A , d_result , d_temp , rawPerThread , TIMES);

    hipMemcpy(h_result , d_result , MATRIX_SIZE , hipMemcpyDeviceToHost);
    printf("first_elm = %f\n",*h_result);

    return 0;
}