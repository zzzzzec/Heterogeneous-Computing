#include "hip/hip_runtime.h"
/*关于此文件的注释*/
#include <stdio.h>
#include <stdlib.h>

__global__  void test(){
    double a = 1.01;
    float b = 1.01;
    for (int i = 0; i < 10; i++)
    {
        a = a * a;
        b = b * b;
    }
    printf("down thread %d \n",threadIdx.x);
    printf("%f \n%f \n",a,b);
}

void main() { 
    uint32_t a = 56;
    uint32_t temp = 0;
    int list[32];
    for (int i = 0; i < 32; i++)
    {
      list[i] = 0;
    }

    for (int i = 0; i < 4*sizeof(uint32_t); i++)
    {   
        if((a>>i)&0xfffffffe){
          printf("v + 1\n");
          list[i] = 1;
        } 
    }   
    for (int i = 0; i < 32; i++)
    {
      printf("%d ", list[i]);
    }
    
}